#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <chrono>
#include "permutate.cuh"


using namespace std;


// definiowanie makra do obs�ugi b��d�w
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    // sprawdzenie czy dzia�anie funkcji zako�czy�o si� b��dem
    if (code != hipSuccess) {
        // printowanie b��du, pliku w kt�rym wyst�pi� oraz linii kodu
        fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            // je�li przekazujemy argument true to exitujemy program
            exit(code);
        }
    }
}



long long factorial(int n) {
    int resoult = 1;
    while (n) resoult *= n--;
    return resoult;
}



void swap(int& a, int& b) {
    int temp = b;
    b = a;
    a = temp;
}

__device__ void swapGPU(int& a, int& b) {
    int temp = b;
    b = a;
    a = temp;
}


int partition(int arr[], int start, int end) {

    int pivot = arr[start];

    int count = 0;
    for (int i = start + 1; i <= end; i++) {
        if (arr[i] <= pivot)
            count++;
    }

    // Giving pivot element its correct position
    int pivotIndex = start + count;
    swap(arr[pivotIndex], arr[start]);

    // Sorting left and right parts of the pivot element
    int i = start, j = end;

    while (i < pivotIndex && j > pivotIndex) {

        while (arr[i] <= pivot) {
            i++;
        }

        while (arr[j] > pivot) {
            j--;
        }

        if (i < pivotIndex && j > pivotIndex) {
            swap(arr[i++], arr[j--]);
        }
    }

    return pivotIndex;
}



__device__ int partitionGPU(int arr[], int start, int end) {

    int pivot = arr[start];

    int count = 0;
    for (int i = start + 1; i <= end; i++) {
        if (arr[i] <= pivot)
            count++;
    }

    // Giving pivot element its correct position
    int pivotIndex = start + count;
    swapGPU(arr[pivotIndex], arr[start]);

    // Sorting left and right parts of the pivot element
    int i = start, j = end;

    while (i < pivotIndex && j > pivotIndex) {

        while (arr[i] <= pivot) {
            i++;
        }

        while (arr[j] > pivot) {
            j--;
        }

        if (i < pivotIndex && j > pivotIndex) {
            swapGPU(arr[i++], arr[j--]);
        }
    }

    return pivotIndex;
}


void quickSort(int arr[], int start, int end) {

    // base case
    if (start >= end)
        return;

    // partitioning the array
    int p = partition(arr, start, end);

    // Sorting the left part
    quickSort(arr, start, p - 1);

    // Sorting the right part
    quickSort(arr, p + 1, end);
}

__device__ void quickSortGPU(int arr[], int start, int end) {

    // base case
    if (start >= end)
        return;

    // partitioning the array
    int p = partitionGPU(arr, start, end);

    // Sorting the left part
    quickSortGPU(arr, start, p - 1);

    // Sorting the right part
    quickSortGPU(arr, p + 1, end);
}



void next_permutation(int n, int current_permutation[]) {
    // set max_index to index of last value
    int max_index = n - 1;

    // iterate over elements in array to find peak
    for (int i = n - 2; i >= 0; i--) {
        // checking if current element is peak
        if (current_permutation[i] < current_permutation[max_index]) {
            // checking if after peak - from right side there are elements smaller 
            // than peak but bigger than element on very left of peak
            // done by iterating and finding element which fulfills conditions
            int min_index = max_index;
            for (int j = i + 2; j <= n - 1; j++) {
                if (current_permutation[j] < current_permutation[min_index] &&
                    current_permutation[j] > current_permutation[i]) {
                    min_index = j;
                }
            }

            // now time to swap two elements - one on the very left of the peak
            // second is defined as min_index
            swap(current_permutation[i], current_permutation[min_index]);

            // if swap took place on elements beyond n-2 there is need to sort those elements 
            if (i < n - 2) {
                quickSort(current_permutation, i + 1, n - 1);
            }
            break;
        }
        else {
            max_index = i;
        }
    }

    /*for (int i = 0; i < n; i++) printf("%d\t", current_permutation[i]);
    printf("\n");*/
}


__global__ void next_permutationGPU(int n, int current_permutation[]) {
    // set max_index to index of last value
    int max_index = n - 1;

    // iterate over elements in array to find peak
    for (int i = n - 2; i >= 0; i--) {
        // checking if current element is peak
        if (current_permutation[i] < current_permutation[max_index]) {
            // checking if after peak - from right side there are elements smaller 
            // than peak but bigger than element on very left of peak
            // done by iterating and finding element which fulfills conditions
            int min_index = max_index;
            for (int j = i + 2; j <= n - 1; j++) {
                if (current_permutation[j] < current_permutation[min_index] &&
                    current_permutation[j] > current_permutation[i]) {
                    min_index = j;
                }
            }

            // now time to swap two elements - one on the very left of the peak
            // second is defined as min_index
            swapGPU(current_permutation[i], current_permutation[min_index]);

            // if swap took place on elements beyond n-2 there is need to sort those elements 
            if (i < n - 2) {
                quickSortGPU(current_permutation, i + 1, n - 1);
            }
            break;
        }
        else {
            max_index = i;
        }
    }

    /*for (int i = 0; i < n; i++) printf("%d\t", current_permutation[i]);
    printf("\n");*/
}


void find_ith_permutation(int arr[], int n, int index) {

    // stworzenie tablicy, na kt�rej b�d� przekszta�cenia
    int* _arr = new int(n);
    for (int j = 0; j < n; j++) {
        _arr[j] = arr[j];
    }
    
    // create array with known size equal to n
    int* factoradic = new int(n);

    // factorial decomposition with modulo function
    int rest = index;
    for (int j = 1; j <= n; j++) {
        factoradic[n - j] = rest % j;
        rest /= j;
    }

    // array to contain target permutation
    int* permutation_arr = new int(n);
    int _n = n - 1;

    // iteration over all elements in factoradic
    for (int j = 0; j < n; j++) {
        // Assigning factoradic[j]-th element of array to target array 
        permutation_arr[j] = _arr[factoradic[j]];

        // instead of creating new array I am moving all elements which will
        // still be in my factoradic to the left and I am decreasing size of
        // this array to assure that I am only using proper part of it
        for (int k = 0; k < (_n - factoradic[j]); k++) {
            swap(_arr[factoradic[j] + k], _arr[factoradic[j] + k + 1]);
        }
        _n--;
    }
    // just simple print of permutation
    for (int o = 0; o < n; o++) {
        // printf("%d\t", permutation_arr[o]);
        cout << permutation_arr[o] << "    ";
    }
    cout << endl;
    // printf("\n");

    return;
}




__global__ void find_ith_permutationGPU(int *sol, int *arr, int n, int sol_num) {

    int tid = threadIdx.x;
    if (tid > sol_num) {
        return;
    }

    // stworzenie tablicy, na kt�rej b�d� przekszta�cenia
    int* _arr = new int(n);
    for (int i = 0; i < n; i++) {
        _arr[i] = arr[i];
    }
       
    // create array with known size equal to n
    int* factoradic = new int(n);

    // factorial decomposition with modulo function
    int rest = tid + 1;
    for (int j = 1; j <= n; j++) {
        factoradic[n - j] = rest % j;
        rest /= j;
    }

    // array to contain target permutation
    int* permutation_arr = new int(n);
    int _n = n - 1;

    // iteration over all elements in factoradic
    for (int j = 0; j < n; j++) {
        // Assigning factoradic[j]-th element of array to target array 
        permutation_arr[j] = _arr[factoradic[j]];

        // instead of creating new array I am moving all elements which will
        // still be in my factoradic to the left and I am decreasing size of
        // this array to assure that I am only using proper part of it
        for (int k = 0; k < (_n - factoradic[j]); k++) {
            swapGPU(_arr[factoradic[j] + k], _arr[factoradic[j] + k + 1]);
        }
        _n--;
    }
    // put proper element into target array
    for (int o = 0; o < n; o++) {
        sol[tid * n + o] = permutation_arr[o];
    }

    return;
}



int main()
{
    int n = 3;
    int solutions_number = factorial(n) - 1;

    int* first_permutation;

    // obliczenie rozmiaru w bajtach tablicy pojedynczej permutacji
    int size_in_bytes = n * sizeof(int);
    
    first_permutation = (int*)malloc(size_in_bytes);

    for (int i = 0; i < n; i++) {
        first_permutation[i] = i + 1;
        printf("%d\t", first_permutation[i]);
    }
    printf("\n");


    auto CPU_start = chrono::high_resolution_clock::now();

    /*for (int o = 0; o < solutions_number; o++) {
        next_permutation(n, first_permutation);
    }*/
    /*for (int o = 1; o <= solutions_number; o++) {
        find_ith_permutation(first_permutation, n, o);
    }*/

    auto CPU_finish = chrono::high_resolution_clock::now();

    auto duration = chrono::duration_cast<chrono::microseconds>(CPU_finish - CPU_start);

    /*int excel_row = 98;
    find_ith_permutation(first_permutation, n, excel_row - 2)*/;

    // ponowne uzupe�nienie first_permutation pierwotnym ci�giem
    for (int i = 0; i < n; i++) {
        first_permutation[i] = i + 1;
    }

    // stworzenie wska�nika na tablic� z permutacj� pocz�tkow�
    int* first_permutationGPU;

    // stworzenie wska�nika na tablic� rozwi�zaniami w GPU oraz w CPU
    int* solutionsGPU;
    int* solutionsCPU;

    // obliczenie rozmiaru w bajtach tablicy rozwi�za� -> ka�de rozwi�zanie to n int�w wi�c n * sizeof(int) * ilo�� rozwi�za�
    int size_in_bytes_of_solutions = n * solutions_number * sizeof(int);
    solutionsCPU = (int*)malloc(size_in_bytes_of_solutions);


    // alokacja pami�ci na GPU oraz na CPU
    gpuErrorCheck(hipMalloc((void**)&first_permutationGPU, size_in_bytes));
    gpuErrorCheck(hipMalloc((void**)&solutionsGPU, size_in_bytes_of_solutions));
    solutionsCPU = (int*)malloc(size_in_bytes_of_solutions);

    // kopiowanie pami�ci z CPU do GPU
    hipMemcpy(first_permutationGPU, first_permutation, size_in_bytes, hipMemcpyHostToDevice);
    hipMemcpy(solutionsGPU, solutionsCPU, size_in_bytes_of_solutions, hipMemcpyHostToDevice);

    // praca ma zosta� wykonana na jedym w�tku - tymczasowe
    dim3 block(1024);
    dim3 grid(1);

    // wywo�anie
    auto GPU_start = chrono::high_resolution_clock::now();
    find_ith_permutationGPU <<< grid, block >>> (solutionsGPU, first_permutationGPU, n, solutions_number);
    
    // odczekanie a� zostanie uko�czone zadanie kernela
    hipDeviceSynchronize();

    auto GPU_finish = chrono::high_resolution_clock::now();
    auto GPUduration = chrono::duration_cast<chrono::microseconds>(GPU_finish - GPU_start);

    // kopiowanie obliczonych danych spowrotem do CPU
    hipMemcpy(solutionsCPU, solutionsGPU, size_in_bytes_of_solutions, hipMemcpyDeviceToHost);

    for (int p = 0; p < solutions_number; p++) {
        for (int r = 0; r < n; r++) {
            printf("%d\t", solutionsCPU[n*p + r]);
        }
        printf("\n");
    }
    // printowanie czas�w oblicze�
    printf("Obliczenia dla %d!\n", n);
    printf("CPU time:\t%d us\n", static_cast<double>(duration.count()));
    printf("GPU time:\t%d us\n", static_cast<double>(GPUduration.count()));


    // zwolnienie pami�ci w GPU
    hipFree(first_permutationGPU);

    hipDeviceReset();

    return 0;
}

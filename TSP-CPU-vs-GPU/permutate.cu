#include "hip/hip_runtime.h"


#include <iostream>
#include <chrono>

//#include <stdio.h>
//#include <algorithm>
//#include "permutate.cuh"
#pragma comment(lib, "winmm.lib")


using namespace std;


// definiowanie makra do obs�ugi b��d�w
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
	// sprawdzenie czy dzia�anie funkcji zako�czy�o si� b��dem
	if (code != hipSuccess) {
		// printowanie b��du, pliku w kt�rym wyst�pi� oraz linii kodu
		fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) {
			// je�li przekazujemy argument true to exitujemy program
			exit(code);
		}
	}
}


unsigned long long factorial(int n) {
	unsigned long long resoult = 1;
	while (n) resoult *= n--;
	return resoult;
}



void swap(int& a, int& b) {
	int temp = b;
	b = a;
	a = temp;
}

__device__ void swapGPU(int& a, int& b) {
	int temp = b;
	b = a;
	a = temp;
}

//
//int partition(int arr[], int start, int end) {
//
//    int pivot = arr[start];
//
//    int count = 0;
//    for (int i = start + 1; i <= end; i++) {
//        if (arr[i] <= pivot)
//            count++;
//    }
//
//    // Giving pivot element its correct position
//    int pivotIndex = start + count;
//    swap(arr[pivotIndex], arr[start]);
//
//    // Sorting left and right parts of the pivot element
//    int i = start, j = end;
//
//    while (i < pivotIndex && j > pivotIndex) {
//
//        while (arr[i] <= pivot) {
//            i++;
//        }
//
//        while (arr[j] > pivot) {
//            j--;
//        }
//
//        if (i < pivotIndex && j > pivotIndex) {
//            swap(arr[i++], arr[j--]);
//        }
//    }
//
//    return pivotIndex;
//}
//
//
//
//__device__ int partitionGPU(int arr[], int start, int end) {
//
//    int pivot = arr[start];
//
//    int count = 0;
//    for (int i = start + 1; i <= end; i++) {
//        if (arr[i] <= pivot)
//            count++;
//    }
//
//    // Giving pivot element its correct position
//    int pivotIndex = start + count;
//    swapGPU(arr[pivotIndex], arr[start]);
//
//    // Sorting left and right parts of the pivot element
//    int i = start, j = end;
//
//    while (i < pivotIndex && j > pivotIndex) {
//
//        while (arr[i] <= pivot) {
//            i++;
//        }
//
//        while (arr[j] > pivot) {
//            j--;
//        }
//
//        if (i < pivotIndex && j > pivotIndex) {
//            swapGPU(arr[i++], arr[j--]);
//        }
//    }
//
//    return pivotIndex;
//}
//
//
//void quickSort(int arr[], int start, int end) {
//
//    // base case
//    if (start >= end)
//        return;
//
//    // partitioning the array
//    int p = partition(arr, start, end);
//
//    // Sorting the left part
//    quickSort(arr, start, p - 1);
//
//    // Sorting the right part
//    quickSort(arr, p + 1, end);
//}
//
//__device__ void quickSortGPU(int arr[], int start, int end) {
//
//    // base case
//    if (start >= end)
//        return;
//
//    // partitioning the array
//    int p = partitionGPU(arr, start, end);
//
//    // Sorting the left part
//    quickSortGPU(arr, start, p - 1);
//
//    // Sorting the right part
//    quickSortGPU(arr, p + 1, end);
//}
//
//
//
//void next_permutation(int n, int current_permutation[]) {
//    // set max_index to index of last value
//    int max_index = n - 1;
//
//    // iterate over elements in array to find peak
//    for (int i = n - 2; i >= 0; i--) {
//        // checking if current element is peak
//        if (current_permutation[i] < current_permutation[max_index]) {
//            // checking if after peak - from right side there are elements smaller 
//            // than peak but bigger than element on very left of peak
//            // done by iterating and finding element which fulfills conditions
//            int min_index = max_index;
//            for (int j = i + 2; j <= n - 1; j++) {
//                if (current_permutation[j] < current_permutation[min_index] &&
//                    current_permutation[j] > current_permutation[i]) {
//                    min_index = j;
//                }
//            }
//
//            // now time to swap two elements - one on the very left of the peak
//            // second is defined as min_index
//            swap(current_permutation[i], current_permutation[min_index]);
//
//            // if swap took place on elements beyond n-2 there is need to sort those elements 
//            if (i < n - 2) {
//                quickSort(current_permutation, i + 1, n - 1);
//            }
//            break;
//        }
//        else {
//            max_index = i;
//        }
//    }
//
//    /*for (int i = 0; i < n; i++) printf("%d\t", current_permutation[i]);
//    printf("\n");*/
//}
//
//
//__global__ void next_permutationGPU(int n, int current_permutation[]) {
//    // set max_index to index of last value
//    int max_index = n - 1;
//
//    // iterate over elements in array to find peak
//    for (int i = n - 2; i >= 0; i--) {
//        // checking if current element is peak
//        if (current_permutation[i] < current_permutation[max_index]) {
//            // checking if after peak - from right side there are elements smaller 
//            // than peak but bigger than element on very left of peak
//            // done by iterating and finding element which fulfills conditions
//            int min_index = max_index;
//            for (int j = i + 2; j <= n - 1; j++) {
//                if (current_permutation[j] < current_permutation[min_index] &&
//                    current_permutation[j] > current_permutation[i]) {
//                    min_index = j;
//                }
//            }
//
//            // now time to swap two elements - one on the very left of the peak
//            // second is defined as min_index
//            swapGPU(current_permutation[i], current_permutation[min_index]);
//
//            // if swap took place on elements beyond n-2 there is need to sort those elements 
//            if (i < n - 2) {
//                quickSortGPU(current_permutation, i + 1, n - 1);
//            }
//            break;
//        }
//        else {
//            max_index = i;
//        }
//    }
//
//    /*for (int i = 0; i < n; i++) printf("%d\t", current_permutation[i]);
//    printf("\n");*/
//}
//


void find_ith_permutation(int arr[], int n, int index, int* sol) {

	// stworzenie tablicy, na kt�rej b�d� przekszta�cenia
	// int* _arr = new int(n);
	int* _arr = (int*)malloc(n * sizeof(int));
	for (int j = 0; j < n; j++) {
		_arr[j] = arr[j];
	}

	// create array with known size equal to n
	// int* factoradic = new int(n);
	int* factoradic = (int*)malloc(n * sizeof(int));

	// factorial decomposition with modulo function
	int rest = index;
	for (int j = 1; j <= n; j++) {
		factoradic[n - j] = rest % j;
		rest /= j;
		// printf("factoradic[%d] = %d\n", n - j, factoradic[n - j]);
	}

	// array to contain target permutation
	// int* permutation_arr = new int(n);
	int* permutation_arr = (int*)malloc(n * sizeof(int));
	int _n = n - 1;

	// iteration over all elements in factoradic
	for (int j = 0; j < n; j++) {
		// Assigning factoradic[j]-th element of array to target array 
		permutation_arr[j] = _arr[factoradic[j]];

		// instead of creating new array I am moving all elements which will
		// still be in my factoradic to the left and I am decreasing size of
		// this array to assure that I am only using proper part of it
		for (int k = 0; k < (_n - factoradic[j]); k++) {
			swap(_arr[factoradic[j] + k], _arr[factoradic[j] + k + 1]);
		}
		_n--;
	}
	// just simple print of permutation
	for (int o = 0; o < n; o++) {
		sol[index * n + o] = permutation_arr[o];
		// printf("%d\t", permutation_arr[o]);
	}
	// printf("\n");

	free(factoradic);
	free(permutation_arr);
	free(_arr);

	return;
}




__global__ void find_ith_permutationGPU(int* sol, int* arr, int n, int sol_num) {

	// calculating id for each thread
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = offset + tid;

	if (gid > sol_num) {
		return;
	}

	// stworzenie tablicy, na kt�rej b�d� przekszta�cenia
	int* _arr = new int(n);
	if (_arr == NULL) {
		printf("Memory not allocated at: _arr \t\t\t gid: %d\n", gid);
	}

	for (int i = 0; i < n; i++) {
		_arr[i] = arr[i];
	}

	// create array with known size equal to n
	int* factoradic = new int(n);

	if (factoradic == NULL) {
		printf("Memory not allocated at: factoradic \t\t gid: %d\n", gid);
	}

	// factorial decomposition with modulo function
	int rest = gid;
	for (int j = 1; j <= n; j++) {
		factoradic[n - j] = rest % j;
		rest /= j;
	}

	// array to contain target permutation
	int* permutation_arr = new int(n);

	if (permutation_arr == NULL) {
		printf("Memory not allocated at: permutation_arr \t gid: %d\n", gid);
	}

	int _n = n - 1;

	// iteration over all elements in factoradic
	for (int j = 0; j < n; j++) {
		// Assigning factoradic[j]-th element of array to target array 
		permutation_arr[j] = _arr[factoradic[j]];

		// instead of creating new array I am moving all elements which will
		// still be in my factoradic to the left and I am decreasing size of
		// this array to assure that I am only using proper part of it
		for (int k = 0; k < (_n - factoradic[j]); k++) {
			swapGPU(_arr[factoradic[j] + k], _arr[factoradic[j] + k + 1]);
		}
		_n--;
	}
	// put proper element into target array
	for (int o = 0; o < n; o++) {
		sol[gid * n + o] = permutation_arr[o];
	}
	delete[] _arr;
	delete[] factoradic;
	delete[] permutation_arr;
	return;
}


bool checkValidity(int* GPU, int* CPU, int sol_num, int n) {
	
	for (int i = 0; i < sol_num; i++) {
		for (int j = 0; j < n; j++) {
			if (GPU[i * n + j] != CPU[i * n + j]) {
				printf("\nNot valid data at index : %d\n", i);
				
				// pokazanie kilku rozwi�za� z miejsca wyst�pienia b��du
				for (int p = i ; p < i + 5; p++) {
					printf("%d\t\t", p);
					for (int r = 0; r < n; r++) {
						printf("%d\t", GPU[n * p + r]);
					}
					printf("\t\t");
					for (int r = 0; r < n; r++) {
						printf("%d \t", CPU[n * p + r]);
					}
					printf("\n");
				}
				return false;
			}
		}
	}
	return true;
}


struct dimensions {
	int block;
	int grid_x;
};


dimensions get_dimensions(int sol_num) {
	dimensions dim;
	if (sol_num < 1024) {
		// polecane warto�ci to 128 / 256 - zawsze wielokrotno�� 32
		int minimal_block_size = 128; 
		
		// how many minimal_block_size fits in sol_num
		int help = sol_num / minimal_block_size;

		// make it minimal value that covers all solutions
		dim.block = minimal_block_size + help * minimal_block_size;
		dim.grid_x = 1;
		return dim;
	}
	int max_threads_per_block = 1024;
	dim.block = max_threads_per_block;
	dim.grid_x = 1 + sol_num / max_threads_per_block;
	return dim;
}


 int main(int argc, char **argv) {
//int main() {

	int n = atoi(argv[1]);
	// int n = 10;

	unsigned long long solutions_number = factorial(n);

	int* first_permutation = 0;

	// obliczenie rozmiaru w bajtach tablicy pojedynczej permutacji
	int size_in_bytes = n * sizeof(int);

	first_permutation = (int*)malloc(static_cast<size_t>(n * sizeof(int)));

	// filling array with values of 0-th permutation
	for (int i = 0; i < n; i++) {
		first_permutation[i] = i + 1;
	}

	// for (int o = 0; o < solutions_number; o++) {
	//     next_permutation(n, first_permutation);
	// }
	// std::printf("CPU computing done!\n");
	// cout << "CPU computing done!\n";
	// 
	// // ponowne uzupe�nienie first_permutation pierwotnym ci�giem
	// for (int i = 0; i < n; i++) {
	// 	first_permutation[i] = i + 1;
	// }

	// obliczenie rozmiaru w bajtach tablicy rozwi�za� -> ka�de rozwi�zanie to n int�w wi�c n * sizeof(int) * ilo�� rozwi�za�
	unsigned long long size_in_bytes_of_solutions = n * solutions_number * sizeof(int);

	// wska�nik na rozwi�zania pochodz�ce z GPU w RAMie
	int* h_solutionsGPU = (int*)malloc(static_cast<size_t>(n * solutions_number * sizeof(int)));
	// int* h_solutionsGPU = new int(static_cast<unsigned long long>(n) * solutions_number);
	if (h_solutionsGPU == NULL) {
		printf("Memory not allocated.\n");
	}

	// alokacja pami�ci na GPU oraz na CPU

	// stworzenie wska�nika na tablic� rozwi�za� z GPU znajduj�c� si� w VRAMie
	int* d_solutionsGPU;
	gpuErrorCheck(hipMalloc((void**)&d_solutionsGPU, static_cast<size_t>(n * solutions_number * sizeof(int))));
	if (d_solutionsGPU == NULL) {
		printf("Memory not allocated.\n");
	}

	// stworzenie wska�nika na tablic� z permutacj� pocz�tkow�
	int* first_permutationGPU;
	hipMalloc((void**)&first_permutationGPU, static_cast<size_t>(n * sizeof(int)));

	// kopiowanie pami�ci z CPU do GPU
	gpuErrorCheck(hipMemcpy(first_permutationGPU, first_permutation, size_in_bytes, hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_solutionsGPU, h_solutionsGPU, size_in_bytes_of_solutions, hipMemcpyHostToDevice));

	// ka�dy blok mo�e mie� maksymalnie 1024 thread-y dlatego daj� tam max warto��
	// dim3 block(1024);

	// grid to zestawienie obok siebie blok�w thread-�w, mo�e by� w osi x maksymalnie 2^32 - 1 thread�w
	// dim3 grid(1024, 1024);

	dimensions dims = get_dimensions(solutions_number);
	dim3 block(dims.block, 1, 1);
	dim3 grid(dims.grid_x, 1, 1);
	// printf("block.x : %d\n", block.x);
	// printf("grid.x : %d\n", grid.x);

	// timer start
	auto GPU_start = chrono::high_resolution_clock::now();

	// calling kernel
	find_ith_permutationGPU <<< grid, block >>> (d_solutionsGPU, first_permutationGPU, n, solutions_number);

	// wait till device sunc
	gpuErrorCheck(hipDeviceSynchronize());

	auto GPU_finish = chrono::high_resolution_clock::now();
	auto GPU_duration = chrono::duration_cast<chrono::microseconds>(GPU_finish - GPU_start);

	// kopiowanie obliczonych danych spowrotem do CPU
	gpuErrorCheck(hipMemcpy(h_solutionsGPU, d_solutionsGPU, size_in_bytes_of_solutions, hipMemcpyDeviceToHost));


	// wska�nik na rozwi�zania z CPU w RAMie
	int* h_solutionsCPU;
	h_solutionsCPU = (int*)malloc(static_cast<size_t>(n * solutions_number * sizeof(int)));

	if (h_solutionsCPU == NULL) {
		printf("Memory not allocated.\n");
	}
	// ----------------------------------TO G�WNO PONI�EJ wed�ug mnie wcze�niej powodowa�o b��dy--------------------------------------
	// https://stackoverflow.com/questions/28289312/illegal-memory-access-on-cudadevicesynchronize  - fajnie opisane co w zasadzie si� odpierdala - wykraczam poza pamie�
	auto CPU_start = chrono::high_resolution_clock::now();
	for (int o = 0; o < solutions_number; o++) {
		find_ith_permutation(first_permutation, n, o + 1, h_solutionsCPU);
	}
	 auto CPU_finish = chrono::high_resolution_clock::now();
	 auto CPU_duration = chrono::duration_cast<chrono::microseconds>(CPU_finish - CPU_start);
	// ------------------------------------------------------------------------------------------------------------------------------

	// filling up solutions with first permutation
	for (int i = 1; i <= n; i++) {
		h_solutionsGPU[i - 1] = i;
		h_solutionsCPU[i - 1] = i;
	}

	/*for (int p = 0; p < solutions_number; p++) {
		printf("%d\t\t", p);
		for (int r = 0; r < n; r++) {
			printf("%d\t", h_solutionsGPU[n * p + r]);
		}
		printf("\t\t");
		for (int r = 0; r < n; r++) {
			printf("%d\t", h_solutionsCPU[n * p + r]);
		}
		printf("\n");
	}*/

	// sprawdzenie czy dane z GPU s� jednokowe jak te z CPU
	bool data_equality = checkValidity(h_solutionsGPU, h_solutionsCPU, solutions_number, n);
	
	// printowanie czas�w oblicze�
	printf("Obliczenia dla %d!\n", n);
	if (data_equality) printf("Obliczenia sa poprawne\n");
	else printf("Niepoprawne obliczenia\n");
	printf("CPU time:\t%lld us\n", CPU_duration.count());
	printf("GPU time:\t%lld us\n", GPU_duration.count());

	//zwolnienie pami�ci w GPU
	hipFree(first_permutationGPU);
	hipFree(d_solutionsGPU);
	free(h_solutionsCPU);
	free(h_solutionsGPU);

	hipDeviceReset();

	return 0;
}
